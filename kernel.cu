#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <iostream>
#include <vector>
#include <iterator>
#include <algorithm>
#include <chrono>

#ifdef __INTELLISENSE__
#include "intellisense_cuda_intrinsics.h"
#define KERNEL_ARGS(grid, block)
#else
#define KERNEL_ARGS(grid, block) <<< grid, block >>>
#endif



typedef struct {
    int id1;
    int id2;
    float similarity;
} Pair;



// Method signatures
__global__ void histDupeKernel(const float*, int, Pair*, int*, int, float*);
hipError_t findDupes(const float*, unsigned int, std::vector<Pair>&, int*, int, float*);



int main(int argc, char* argv[]) {

    if (argc < 2) {
        fprintf(stderr, "Too few arguments. Expected 1\n\nUsage: %s DATA_PATH\n", argv[0]);
        return 1;
    }


    // Initialize variables
    int max_results = 1000000;
    float confidence = 0.99f;
    float color_variance = 0.25f;
    int N = 50000;
    bool cuda = true;

    std::chrono::steady_clock::time_point time;


    // Print some diagnostics
    std::cout << "Datafile Path: " << argv[1] << std::endl;
    std::cout << "N: " << N << std::endl;
    std::cout << "Max Results: " << max_results << std::endl;
    std::cout << "Confidence: " << confidence << std::endl;
    std::cout << "Color Variance: " << color_variance << std::endl;


    // Allocate some arrays
    std::cout << "Allocating memory..." << std::endl;
    time = std::chrono::steady_clock::now();
    int* ids = new int[N]; // Mapping of actual index to ID of histogram
    float* data = new float[128 * N]; // End-to-end array of all histograms. Each histogram consists of 128 floats
    float* conf = new float[N]; // Confidence array; allows using stricter confidence for black and white images
    std::vector<Pair> pairs; // Vector of similar pairs (to be populated)
    std::cout << "Allocated memory in: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - time).count() << " ms" << std::endl;



    // Read test data from file
    std::cout << "Reading data from file: " << argv[1] << "..." << std::endl;
    time = std::chrono::steady_clock::now();

    FILE* file;
    file = fopen(argv[1], "r");

    for (int i = 0; i < N; i++) {
        fscanf(file, "%d", &ids[i]);
        for (int j = 0; j < 128; j++) {
            fscanf(file, "%f", &data[i * 128 + j]);
        }
    }

    fclose(file);
    std::cout << "Read data in: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - time).count() << " ms" << std::endl;



    // Build confidence array
    std::cout << "Building confidence array..." << std::endl;
    time = std::chrono::steady_clock::now();
    float confidence_square = 1 - (1 - confidence) * (1 - confidence);
    for (int i = 0; i < N; i++) {
        float d = 0;

        // Compute sum of color variance across histogram
        for (int k = 0; k < 32; k++) {
            float r = data[i * 128 + k + 32];
            float g = data[i * 128 + k + 64];
            float b = data[i * 128 + k + 96];
            d += __max(__max(r, g), b) - __min(__min(r, g), b);
        }

        if (d > color_variance) {
            conf[i] = confidence; // Image is colorful, use normal confidence
        } else {
            conf[i] = confidence_square; // Image is not colorful, use squared confidence
        }
    }
    std::cout << "Built confidence array in: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - time).count() << " ms" << std::endl;


    // Find duplicates
    std::cout << "Finding duplicates..." << std::endl;
    hipError_t cudaStatus;
    int result_count = 0;
    time = std::chrono::steady_clock::now();
    if (cuda) {
        // With CUDA
        cudaStatus = findDupes(data, N, pairs, &result_count, max_results, conf);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Kernel failed!");
            return 1;
        }
        for (int i = 0; i < result_count; i++) {
            pairs[i].id1 = ids[pairs[i].id1];
            pairs[i].id2 = ids[pairs[i].id2];
        }
    } else {
        // Sequentially
        for (int i = 0; i < N; i++) {
            for (int j = i + 1; j < N; j++) {
                double d = 0;
                for (int k = 0; k < 128; k++) {
                    d += fabs(data[i * 128 + k] - data[j * 128 + k]);
                }
                d = 1 - (d / 8);
                if (d > fmaxf(conf[i], conf[j])) {
                    Pair p;
                    p.similarity = (float) d;
                    p.id1 = ids[i];
                    p.id2 = ids[j];
                    pairs.push_back(p);
                    result_count++;
                }
            }
        }
    }
    std::cout << "Found duplicates in: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - time).count() << " ms" << std::endl;


    // Print some results
    std::cout << "Found pairs: " << result_count << std::endl;
    std::cout << "Example results:" << std::endl;
    for (int i = 0; i < __min(result_count, 10); i++) {
        std::cout << "\t" << ids[pairs[i].id1] << " - " << ids[pairs[i].id2] << ":\t\t" << pairs[i].similarity << std::endl;
    }


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    // Delete arrays
    delete[] data;
    delete[] conf;
    delete[] ids;

    return 0;
}



hipError_t findDupes(const float* data, unsigned int N, std::vector<Pair>& pairs, int* result_count, int max_results, float* confidence) {

    float* d_data; // Data device pointer
    Pair* d_pairs; // Pairs device pointer
    float* d_confidence; // Confidence device pointer
    int* d_result_count; // Result count device pointer

    hipError_t cudaStatus; // CUDA error

    std::chrono::steady_clock::time_point time; // Time tracking

    unsigned int dN = N; // Padded device N to match block size
    if (N % 64 != 0) {
        dN = (unsigned int) ceil((double) N / 64) * 64;
    }
    std::cout << "Adjusted N: " << dN << std::endl;

    // Choose which GPU to run on, change this on a multi-GPU system
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    // Allocate GPU buffers
    std::cout << "Allocating GPU memory..." << std::endl;
    time = std::chrono::steady_clock::now();
    cudaStatus = hipMalloc((void**) &d_data, sizeof(float) * 128 * dN);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**) &d_pairs, sizeof(Pair) * max_results);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**) &d_result_count, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**) &d_confidence, sizeof(float) * dN);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    std::cout << "Allocated GPU memory in: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - time).count() << " ms" << std::endl;

    // Copy input data from host memory to GPU buffers
    std::cout << "Copying data to device..." << std::endl;
    time = std::chrono::steady_clock::now();
    cudaStatus = hipMemcpy(d_data, data, sizeof(int) * 128 * N, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(d_confidence, confidence, sizeof(float) * N, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    if (dN > N) {
        float* temp_conf = new float[dN - N];
        for (int i = 0; i < dN - N; i++) temp_conf[i] = 2; // Impossible confidence
        cudaStatus = hipMemcpy(d_confidence + N, temp_conf, sizeof(float) * (dN - N), hipMemcpyHostToDevice);
        delete[] temp_conf;
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
    }
    std::cout << "Copied data to GPU memory in: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - time).count() << " ms" << std::endl;


    // Launch a kernel on the GPU
    std::cout << "Launching kernel..." << std::endl;
    time = std::chrono::steady_clock::now();
    histDupeKernel KERNEL_ARGS((int) ceil((double) N / 64), 64) (d_data, N, d_pairs, d_result_count, max_results, d_confidence);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
        goto Error;
    }
    std::cout << "Ran GPU kernel in: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - time).count() << " ms" << std::endl;

    // Copy output from GPU buffer to host memory.
    std::cout << "Copying results from device..." << std::endl;
    time = std::chrono::steady_clock::now();
    cudaStatus = hipMemcpy((void*) result_count, d_result_count, sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    result_count[0] = __min(result_count[0], max_results); // Clamp result_count
    // Read result pairs into buffer
    Pair* temp_pairs = new Pair[result_count[0]];
    cudaStatus = hipMemcpy((void*) temp_pairs, d_pairs, sizeof(Pair) * result_count[0], hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    // Only keep pairs that are unique (pairs are commutative)
    for (int i = 0; i < result_count[0]; i++) {
        int p1id1 = temp_pairs[i].id1;
        int p1id2 = temp_pairs[i].id2;
        bool found = false;
        for each (const Pair p2 in pairs) {
            if ((p1id1 == p2.id1 && p1id2 == p2.id2) || (p1id1 == p2.id2 && p1id2 == p2.id1)) {
                found = true;
                break;
            }
        }

        if (!found) {
            pairs.push_back(temp_pairs[i]);
        }
    }
    delete[] temp_pairs;
    result_count[0] = (int) pairs.size();
    std::cout << "Retrieved results from GPU memory in: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - time).count() << " ms" << std::endl;

Error:

    // Free cuda memory
    std::cout << "Freeing GPU memory..." << std::endl;
    time = std::chrono::steady_clock::now();
    hipFree(d_data);
    hipFree(d_pairs);
    hipFree(d_result_count);
    hipFree(d_confidence);
    std::cout << "Freed GPU memory in: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - time).count() << " ms" << std::endl;

    return cudaStatus;
}



__global__ void histDupeKernel(const float* data, int N, Pair* results, int* result_count, int max_results, float* confidence) {

    int thread = threadIdx.x; // Thread index within block
    int block = blockIdx.x; // Block index
    int block_size = blockDim.x; // Size of each block

    int block_start = block_size * block;
    int index = block_start + thread; // Index of histogram for this thread

    __shared__ float conf[64]; // Shared array of confidence values for all histograms owned by this block
    conf[thread] = confidence[index]; // Coalesced read of confidence values

    __shared__ float hists[128 * 64];
    for (int i = 0; i < 64; i++) {
        hists[i * 128 + thread] = data[(block_start + i) * 128 + thread];
        hists[i * 128 + thread + 64] = data[(block_start + i) * 128 + 64 + thread];
    }

    __shared__ float other[128]; // Histogram to compare all owned histograms against parallely
    for (int i = 0; i < N && *result_count < max_results; i++) {

        float other_conf = confidence[i]; // All threads read confidence for other histogram into register

        other[thread] = data[i * 128 + thread]; // Coalesced read of other histogram into shared memory
        other[thread + 64] = data[i * 128 + thread + 64];

        __syncthreads(); // Ensure all values read

        if (index < N) {
            float d = 0;
            for (int k = 0; k < 128; k++) { // Compute sum of distances between thread-owned histogram and shared histogram
                d += std::fabsf(hists[thread * 128 + k] - other[k]);
            }
            d = 1 - (d / 8); // Massage the difference into a nice % similarity number, between 0 and 1

            if (i != index && d > fmaxf(conf[thread], other_conf)) { // Don't compare against self, only compare using highest confidence
                int result_index = atomicAdd(result_count, 1); // Increment result count by one atomically
                if (result_index < max_results) {
                    // Store resulting pair
                    results[result_index].similarity = d;
                    results[result_index].id1 = index;
                    results[result_index].id2 = i;
                }
            }
        }

        __syncthreads(); // Ensure all threads have finished before looping and reading new shared histogram
    }

}
